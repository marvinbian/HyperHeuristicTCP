#include "hip/hip_runtime.h"
/* 
   For fitness calcualte in pso algorithm 
   the parallel version.
 */


#include "pso_config.h"
#include <iomanip>
//#include "cal_fitness_kernel.h"
#include <hip/hip_runtime.h>

using namespace std;
/*
__global__ void apsc_parallel(bool *d_CMatrix, int *d_ind, int *d_temp_cover,
        bool *d_max_cover, int num_testcase)
{
    int bid = blockIdx.y * gridDim.x + blockIdx.x;  // y:100  x:block_num 
    int bdim = blockDim.x;                          // 512
    int tid = threadIdx.x;                          // 0 - 511

    __shared__ bool statement[512]; //Record the max_coverage array
    statement[tid] = d_max_cover[blockIdx.x * bdim + tid];
    __syncthreads();
    
    //size_t in_size = (MAXGENE + 512)/512;
    __shared__ int individuals[MAXGENE + 512]; //Record the individuals
    for (int i = 0; i < (MAXGENE+512)/512; i++)
    {
        individuals[i * 512 + tid] = d_ind[blockIdx.y * MAXGENE + i * 512 + tid];
    }
    __syncthreads();

    
    __shared__ int record[512];
    record[tid] = 0;
    __syncthreads();
    
    if (statement[tid])
    {
        int line_num = blockIdx.x * bdim + tid;

        for (int i = 0; i < num_testcase; i++)
        {
            if (d_CMatrix[individuals[i] * CODELINE + line_num])
            {
                record[tid] = i + 1;
                break;
            }
        }
    }
    __syncthreads();
    
    d_temp_cover[bid * bdim + tid] = record[tid];
    //d_temp_cover[bid * bdim +tid] = statement[tid];
    __syncthreads();
}

__global__ void cal_apsc(int *d_temp_cover, float *d_apsc, int *d_effnum, int
        block_num, int num_testcase)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;

    __shared__ int total_cover[512];
    total_cover[tid] = 0;
    __shared__ int max_eff[512];
    max_eff[tid] = 0;
    
    __syncthreads();

    for (int i = 0; i < block_num; i++)
    {
        int pos = bid * block_num * 512 + i * 512 + tid;
        total_cover[tid] += d_temp_cover[pos];
        if (max_eff[tid] < d_temp_cover[pos])
        {
            max_eff[tid] = d_temp_cover[pos];
        }
    }
    __syncthreads();

    for (int i = 256; i > 0; i = i/2)
    {
        if (tid < i)
        {
            total_cover[tid] += total_cover[tid + i];
            if (max_eff[tid] < max_eff[tid + i])
            {
                max_eff[tid] = max_eff[tid + i];
            }
        }
    }
    __syncthreads();

    if (tid == 0)
    {
        d_apsc[bid] = 1.0f - ((float)total_cover[0] / (float)(CODELINE *
                    num_testcase)) + 1.0f / (float)(2 * num_testcase);
        d_effnum[bid] = max_eff[0];
        //d_effnum[bid] = total_cover[0];

    }
    __syncthreads();
}

__global__ void cal_efftime(int *d_ind, int *d_effnum, float *d_TVector,
        int num_testcase, float *d_exetime)
{
    int tid = threadIdx.x;

    extern __shared__ float tvector[];

    int limit = (num_testcase + 512)/512;
    for (int i = 0; i < limit; i++)
    {
        int pos = i * 512 + tid;
        if (pos < num_testcase)
        {
            tvector[pos] = d_TVector[pos];
        }
    }

    __shared__ int loop_limit[SWARMSIZE];
    __shared__ float total_time[SWARMSIZE];
    if (tid < SWARMSIZE)
    {
        loop_limit[tid] = d_effnum[tid];
        total_time[tid] = 0.0f;
    }
    __syncthreads();

    if (tid < SWARMSIZE)
    {
        for (int i = 0; i < loop_limit[tid]; i++)
        {
            int tc_num = d_ind[tid * MAXGENE + i];
            total_time[tid] += tvector[tc_num];
        }
    }
    __syncthreads();

    if (tid < SWARMSIZE)
    {
        d_exetime[tid] = total_time[tid];
    }
    __syncthreads();
    
}

__global__ void cal_front(float *d_apsc, float *d_exetime, bool *d_efficient)
{
    int tid = threadIdx.x;
    
    __shared__ float apsc[SWARMSIZE];
    apsc[tid] = d_apsc[tid];

    __shared__ float exetime[SWARMSIZE];
    exetime[tid] = d_exetime[tid];

    __shared__ bool efficient[SWARMSIZE];
    efficient[tid] = 1;

    __syncthreads();

    for (int i = 0; i < SWARMSIZE; i++)
    {
        if (apsc[tid] < apsc[i] && exetime[tid] > exetime[i])
        {
            efficient[tid] = 0;
            break;
        }
    }

    d_efficient[tid] = efficient[tid];

}
*/// bytest

void cal_apsc_serial(bool *CMatrix, int *particle, float *apsc, int *eff_num, 
        bool *max_cover, int num_testcase)
{
    for (int i = 0; i < SWARMSIZE; i++)
    {
        int sum_testnum = 0;
        int max_testnum = 0;

        for (int j = 0; j < CODELINE; j++)
        {
            if (max_cover[j] == 0) continue;

            for (int k = 0; k < num_testcase; k++)
            {
                bool temp = CMatrix[particle[i * MAXGENE + k] * 
                    CODELINE + j]; 
                
                if (temp == 1)
                {
                    k += 1;
                    sum_testnum += k; 
                    if (k > max_testnum) max_testnum = k;
                    break;
                }
            }
        }
        
        eff_num[i] = max_testnum;
        apsc[i] = 1 - ((float)sum_testnum / (float)(CODELINE * num_testcase))
            + 1.0f / (float)(2 * num_testcase);
    }
}

void cal_efftime_serial(int *particle, int *effnum, float *TVector, float
        *exetime)
{
    for (int i = 0; i < SWARMSIZE; i++)
    {
        float sum_time = 0.0f;
        for (int j = 0; j < effnum[i]; j++)
        {
            sum_time += TVector[particle[j]];       
        }
        exetime[i] = sum_time;
    }
}

void cal_front_serial(float *apsc, float *exetime, bool *efficient, 
        int *eff_mark)
{
    for (int i = 0; i < SWARMSIZE; i++) efficient[i] = 1;
    //for (int i = 0; i < SWARMSIZE; i++) eff_mark[i] = 0;

    for (int i = 0; i < SWARMSIZE; i++)
    {
        for (int j = i + 1; j < SWARMSIZE; j++)
        {
            if (efficient[j] == 0) continue;

            if (apsc[i] < apsc[j] && exetime[i] > exetime[j])
            {
                efficient[i] = 0;
                break;
            }
            else if (apsc[i] > apsc[j] && exetime[i] < exetime[j])
            {
                efficient[j] = 0;
            }
        }
    }
}

extern "C" void Cal_fitness(Swarms* sw_pt, int num_testcase, bool* CMatrix, float*
        TVector, bool* max_cover)
{
    //size_t ind_size = SWARMSIZE * MAXGENE;

    //int *d_ind = NULL;
    //CUDA_CALL(hipMalloc((void**)&d_ind, sizeof(int) * ind_size));
    //CUDA_CALL(hipMemcpy(d_ind, &(sw_pt->particle[0]), sizeof(int) * ind_size,
    //            hipMemcpyHostToDevice));
    
    //float *d_apsc = NULL;
    //int *d_effnum = NULL;
    //float *d_exetime = NULL;
    //bool *d_efficient = NULL;
    //CUDA_CALL(hipMalloc((void**)&d_apsc, sizeof(float) * SWARMSIZE));
    //CUDA_CALL(hipMalloc((void**)&d_effnum, sizeof(int) * SWARMSIZE));
    //CUDA_CALL(hipMalloc((void**)&d_exetime, sizeof(float) * SWARMSIZE));
    //CUDA_CALL(hipMalloc((void**)&d_efficient, sizeof(bool) * SWARMSIZE));
    //CUDA_CALL(hipMemset(d_efficient, 0, sizeof(bool) * SWARMSIZE));

    //int block_num = (CODELINE + 512)/512;
    //cout << "Block Number: " << block_num << endl; 
    //cout << "Test case NO: " << num_testcase << endl;

    //int *d_temp_cover = NULL;
    //CUDA_CALL(hipMalloc((void**)&d_temp_cover, sizeof(int) * SWARMSIZE *
    //            block_num * 512));
    //CUDA_CALL(hipMemset(d_temp_cover, 0, sizeof(int) * SWARMSIZE * block_num *
    //            512));

    //dim3 grid(block_num, SWARMSIZE);
    
    /*
    apsc_parallel<<<grid, 512>>>(d_CMatrix, d_ind, d_temp_cover, d_max_cover,
            num_testcase);
    
    cal_apsc<<<SWARMSIZE, 512>>>(d_temp_cover, d_apsc, d_effnum, block_num,
            num_testcase);
    */

    hipEvent_t fstime, fetime;

    CUDA_CALL(hipEventCreate(&fstime, 0));
    CUDA_CALL(hipEventCreate(&fetime, 0));

    hipEventRecord(fstime, 0);

    cal_apsc_serial(CMatrix, &(sw_pt->particle[0]), &(sw_pt->apsc[0]),
            &(sw_pt->eff_num[0]), max_cover, num_testcase);

    hipEventRecord(fetime, 0);
    hipEventSynchronize(fetime);
    float cptime = 0.0f;
    hipEventElapsedTime(&cptime, fstime, fetime);
    //cout << "   cal_apsc time: " << cptime << endl;
    //size_t tv_size = num_testcase * sizeof(float);
    //cal_efftime<<<1, 512, tv_size>>>(d_ind, d_effnum, d_TVector, num_testcase,
    //        d_exetime);
    hipEventRecord(fstime, 0);

    cal_efftime_serial(&(sw_pt->particle[0]), &(sw_pt->eff_num[0]), TVector, 
            &(sw_pt->exetime[0]));

    hipEventRecord(fetime, 0);
    hipEventSynchronize(fetime);
    cptime = 0.0f;
    hipEventElapsedTime(&cptime, fstime, fetime);
    //cout << "   cal_efftime time: " << cptime << endl;
    //CUDA_CALL(hipMemcpy(&(sw_pt->apsc[0]), d_apsc, sizeof(float) * SWARMSIZE,
    //            hipMemcpyDeviceToHost));
    //CUDA_CALL(hipMemcpy(&(sw_pt->eff_num[0]), d_effnum, sizeof(int) * SWARMSIZE,
    //            hipMemcpyDeviceToHost));
    //CUDA_CALL(hipMemcpy(&(sw_pt->exetime[0]), d_exetime, sizeof(float) *
    //            SWARMSIZE, hipMemcpyDeviceToHost));
    
    //Update the particle best with 1/2 rate 
    /*
    for (int i = 0; i < SWARMSIZE; i++)
    {
        if (sw_pt->apsc[i] > sw_pt->papsc[i] && sw_pt->exetime[i] <
                sw_pt->pexetime[i])
        {
            for (int j = 0; j < MAXGENE; j ++)
            {
                sw_pt->pbest[i * MAXGENE + j] = sw_pt->particle[i * MAXGENE +
                    j];
            }

            sw_pt->papsc[i] = sw_pt->apsc[i];
            sw_pt->pexetime[i] = sw_pt->exetime[i];
        }
        else if (sw_pt->apsc[i] <= sw_pt->papsc[i] && sw_pt->exetime[i] >=
                sw_pt->pexetime[i])
        {
            continue;
        }
        else
        {
            srand((unsigned int)time(0) * i);
            int rand_num = rand()%2;
            if (rand_num)
            {
                for (int j = 0; j < MAXGENE; j++)
                {
                    sw_pt->pbest[i * MAXGENE + j] = sw_pt->particle[i * MAXGENE +
                        j];
                }
                sw_pt->papsc[i] = sw_pt->apsc[i];
                sw_pt->pexetime[i] = sw_pt->exetime[i];
            }
            else continue;
        }
    }
    */
    //cal_front<<<1, SWARMSIZE>>>(d_apsc, d_exetime, d_efficient);
    hipEventRecord(fstime, 0);

    int eff_mark[SWARMSIZE];
    cal_front_serial(&(sw_pt->apsc[0]), &(sw_pt->exetime[0]), 
            &(sw_pt->efficient[0]), &(eff_mark[0])); 
  
    hipEventRecord(fetime, 0);
    hipEventSynchronize(fetime);
    cptime = 0.0f;
    hipEventElapsedTime(&cptime, fstime, fetime);
    //cout << "   cal_front time: " << cptime << endl;
    //CUDA_CALL(hipMemcpy(&(sw_pt->efficient[0]), d_efficient, sizeof(bool) *
    //            SWARMSIZE, hipMemcpyDeviceToHost));
    
    // Update the front set

    hipEventRecord(fstime, 0);

    int eff_num[SWARMSIZE];
    int count = 0;
    for (int i = 0; i < SWARMSIZE; i++)
    {
        if (sw_pt->efficient[i] == 1)
        {
            eff_num[count] = i;
            count++;
        }
    }
   
    bool *mark = NULL;
    mark = (bool*)malloc(sizeof(bool) * count);
    for (int i = 0; i < count; i++) mark[i] = 1;
    
    //testby---------------------------------------
    /* 
    sw_pt->EAapsc[0] = 0.990;
    sw_pt->EAexetime[0] = 90.09;
    sw_pt->EAefficient[0] = 1;
    
    sw_pt->EAapsc[5] = 0.995;
    sw_pt->EAexetime[5] = 100.09;
    sw_pt->EAefficient[5] = 1;
    
    sw_pt->EAlen = 6;
    */

    for (int i = 0; i <= sw_pt->EAlen; i++)
    {
        if (sw_pt->EAefficient[i] == 0)
        {
            continue;
        }
        for (int j = 0; j < count; j++)
        {
            if ((sw_pt->EAapsc[i] > sw_pt->apsc[eff_num[j]]) && (sw_pt->EAexetime[i]
                    < sw_pt->exetime[eff_num[j]]))
            {
                mark[j] = 0;
            }
            else if ((sw_pt->EAapsc[i] < sw_pt->apsc[eff_num[j]]) && (sw_pt->EAexetime[i]
                    > sw_pt->exetime[eff_num[j]]))
            {
                sw_pt->EAefficient[i] = 0;
            }
            else continue;
        }
    }

    //Output------------------------------
    /*
    cout << "count: " << count << endl;
    for (int i = 0; i < count; i++)
    {
        cout << "M: " << mark[i] << endl;
        cout << sw_pt->apsc[eff_num[i]] << endl;
        cout << sw_pt->exetime[eff_num[i]] << endl << endl;
    }
    */

    int front_tail = 0;
    for (int i = 0; i < count; i++)
    {
        if (mark[i] == 0) continue;
        for (int j = front_tail; j < ARCHSIZE; j++)
        {
            if (sw_pt->EAefficient[j] == 0)
            {
                for (int k = 0; k < num_testcase; k++)
                {
                    sw_pt->EA[j * MAXGENE + k] = sw_pt->particle[eff_num[i] *
                        MAXGENE + k];
                }
                sw_pt->EAapsc[j] = sw_pt->apsc[eff_num[i]];
                sw_pt->EAexetime[j] = sw_pt->exetime[eff_num[i]];
                sw_pt->EAefficient[j] = 1;
                front_tail = j + 1;

                if (front_tail > sw_pt->EAlen)
                {
                    sw_pt->EAlen = front_tail;
                }
                break;
            }
            
        }
    }
    
    hipEventRecord(fetime, 0);
    hipEventSynchronize(fetime);
    cptime = 0.0f;
    hipEventElapsedTime(&cptime, fstime, fetime);
    //cout << "   update front time: " << cptime << endl;
    //cout << "Lens: " << sw_pt->EAlen << endl;
    
    //Update the front set end
    /*    
    //The temperate parameters to test the parallel programs.
    //It can leave here untial the whole sure to be finished.

    int *temp_cover = NULL;
    temp_cover = (int*)malloc(sizeof(int) * SWARMSIZE * block_num * 512);
    CUDA_CALL(hipMemcpy(temp_cover, d_temp_cover, sizeof(int) * SWARMSIZE *
                block_num * 512, hipMemcpyDeviceToHost));
    
    float *temp_total = NULL;
    temp_total = (float*)malloc(sizeof(float) * SWARMSIZE);
    CUDA_CALL(hipMemcpy(temp_total, d_exetime, sizeof(float) * SWARMSIZE, 
                hipMemcpyDeviceToHost));
    int *temp_eff = NULL;
    temp_eff = (int*)malloc(sizeof(int) * SWARMSIZE);
    CUDA_CALL(hipMemcpy(temp_eff, d_effnum, sizeof(int) * SWARMSIZE, 
                hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(temp_cover));
    CUDA_CALL(hipFree(temp_total));
    CUDA_CALL(hipFree(temp_eff));
    */    

    //free(temp_cover);
    /*
    CUDA_CALL(hipFree(d_ind));
    CUDA_CALL(hipFree(d_apsc));
    CUDA_CALL(hipFree(d_exetime));
    CUDA_CALL(hipFree(d_efficient));
    CUDA_CALL(hipFree(d_temp_cover));
    */
}
