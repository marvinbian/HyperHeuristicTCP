#include "hip/hip_runtime.h"
/*
    For the PSO crossover operation
    on GPU platform, Including the pbest and gbest, and 
    the gbest is been selected from the front set randomly.
 */

#include <iostream>
#include <hip/hip_runtime.h>

#include "pso_config.h"

using namespace std;

/*
__global__ void crossover_g_singlepoint(int *d_purpose, int *d_element1, int *d_element2,
        bool *d_geffic, bool *d_mark, int *d_rand, int num_testcase, int g_num)
{
    int tid = threadIdx.x; //100
    
    int pos = d_rand[tid];
    int g_count = pos % g_num;
    int gbest_num = 0;

    if (d_geffic[g_count] == 1)
    {
        gbest_num = g_count;
    }
    else
    {
        for (int i = 1; i < g_num; i++)
        {
            if (g_count + i >= g_num)
            {
                g_count = 0;
                i = 0;
                continue;
            }
            else if (d_geffic[g_count + i] == 1)
            {
                gbest_num = g_count + i;
                break;
            }
        }
    }
    //d_purpose[tid] = gbest_num; 
        
    for (int i = 0; i < pos; i++)
    {
        d_purpose[tid * MAXGENE + i] = d_element1[tid * MAXGENE + i];
        d_mark[tid * MAXGENE + d_element1[tid * MAXGENE + i]] = 1;
    }

    for (int i = 0; i < num_testcase; i++)
    {
        if (d_mark[tid * MAXGENE + d_element2[gbest_num * MAXGENE + i]] == 0)
        {
            d_purpose[tid * MAXGENE + pos] = d_element2[gbest_num * MAXGENE 
                + i];
            pos += 1;
            //d_mark[tid * MAXGENE + d_element2[gbest_num * MAXGENE + i]] = 1; //***************
        }
    }
}

__global__ void crossover_singlepoint(int *d_purpose, int *d_element1, int *d_element2,
        bool *d_mark, int *d_rand, int num_testcase)
{
    int tid = threadIdx.x; //100
    
    int pos = d_rand[tid];
        
    for (int i = 0; i < pos; i++)
    {
        d_purpose[tid * MAXGENE + i] = d_element1[tid * MAXGENE + i];
        d_mark[tid * MAXGENE + d_element1[tid * MAXGENE + i]] = 1;
    }

    for (int i = 0; i < num_testcase; i++)
    {
        if (d_mark[tid * MAXGENE + d_element2[tid * MAXGENE + i]] == 0)
        {
            d_purpose[tid * MAXGENE + pos] = d_element2[tid * MAXGENE + i];
            pos += 1;
            //d_mark[tid * MAXGENE + d_element2[tid * MAXGENE + i]] = 1; //***************
        }
    }
}

__global__ void cal_distance_order(int *d_order, float *d_apsc, float *d_exetime,
        float max_apsc, float min_apsc, float max_exetime, float min_exetime)
{
    int tid = threadIdx.x; //100
    
    __shared__ float d_max_apsc;
    __shared__ float d_min_apsc;// = min_apsc;
    __shared__ float d_max_exetime;// = max_exetime;
    __shared__ float d_min_exetime;// = min_exetime;
    
    if (tid == 0) 
    {
        d_max_apsc = max_apsc;
        d_min_apsc = min_apsc;
        d_max_exetime = max_exetime;
        d_min_exetime = min_exetime;
    }
    __syncthreads();

    __shared__ float d_distance[SWARMSIZE];
    d_distance[tid] = 0;

    d_distance[tid] = ((d_apsc[tid] - d_min_apsc) / (d_max_apsc - d_min_apsc)) 
        + ((d_exetime[tid] - d_min_exetime) / (d_max_exetime - d_min_exetime));
    
    __syncthreads();
    
    __shared__ int d_count[SWARMSIZE];
    d_count[tid] = 0; __syncthreads();

    for (int i = 0; i < SWARMSIZE; i++)
    {
        if (tid == i) continue;
        if (d_distance[i] > d_distance[tid])
        {
            d_count[tid] += 1;
        }

    }
    __syncthreads();

    d_order[d_count[tid]] = tid;
}

__global__ void copy_gbest(int *d_rand, int *d_gbest, int *d_pg, bool *d_geffic,
            int num_testcase, int g_num)
{
    //int bid = blockIdx.y * gridDim.x + blockIdx.x; // 50 * 6 + 6
    int tid = threadIdx.x; //50

    __shared__ int g_count[50];
    __shared__ int gbest_num[50];


    
    g_count[tid] = d_rand[tid] % g_num;
    gbest_num[tid] = 0;

    
    if (d_geffic[g_count[tid]] == 1)
    {
        gbest_num[tid] = g_count[tid];
    }
    else
    {
        for (int i = 1; i < g_num; i++)
        {
            if (g_count[tid] + i >= g_num)
            {
                g_count[tid] = 0;
                i = 0;
                continue;
            }
            else if (d_geffic[g_count[tid] + i] == 1)
            {
                gbest_num[tid] = g_count[tid] + i;
                break;

            }
        }
    }
    
    __syncthreads();

    for (int i = 0; i < num_testcase; i++)
    {
        d_pg[tid * MAXGENE + i] = d_gbest[gbest_num[tid] * MAXGENE + i];
    }

    //d_pg[blockIdx.x] = g_count;
    //d_pg[bid * blockDim.x + tid] = d_gbest[(gbest_num * gridDim.x + blockIdx.x)
    //    * blockDim.x + tid];

}

__global__ void copy_particle(int *d_order, int *d_particle, int *d_pg,
            int num_testcase)
{
    //int bid = blockIdx.y * gridDim.x + blockIdx.x; // 6 * 50
    int tid = threadIdx.x; // 50

    //__shared__ int p_pos;
    //if (tid == 0) p_pos = d_order[blockIdx.y];
    //__syncthreads();

    for (int i = 0; i < num_testcase; i++)
    {
        d_pg[(tid + SWARMSIZE / 2) * MAXGENE + i] = d_particle[d_order[tid] *
            MAXGENE + i];
    }
    //d_pg[((blockIdx.y + SWARMSIZE / 2) * gridDim.x + blockIdx.x) * blockDim.x +
    //    tid] = d_particle[(p_pos * gridDim.x + blockIdx.x) * blockDim.x + tid];
}

__global__ void ea_crossover(int *d_rand_cross, int *d_pg, int num_rand)
{
    int bid = blockIdx.x;  //100 
    int tid = threadIdx.x; //304

    extern __shared__ int pos[];
    pos[tid] = d_rand_cross[bid * num_rand * 2 + tid];
    pos[tid + num_rand] = d_rand_cross[bid * num_rand * 2 + tid + num_rand];
    __syncthreads();

    //d_pg[bid * num_rand * 2 + tid] = pos[tid];
    //d_pg[bid * num_rand * 2 + tid + num_rand] = pos[tid + num_rand];
    
    int temp_gen = d_pg[bid * MAXGENE + pos[tid]];
    d_pg[bid * MAXGENE + pos[tid]] = d_pg[bid * MAXGENE + pos[tid + num_rand]];
    d_pg[bid * MAXGENE + pos[tid + num_rand]] = temp_gen; 
}
*///bytest

void distance_order_serial(int *order, float *apsc, float *exetime, float max_apsc,
            float min_apsc, float max_exetime, float min_exetime)
{
    float distance[SWARMSIZE];
    float temp_apsc = max_apsc - min_apsc;
    float temp_exetime = max_exetime - min_exetime;
    
    for (int i = 0; i < SWARMSIZE; i++)
    {
        distance[i] = ((apsc[i] - min_apsc) / temp_apsc) + ((exetime[i] -
                    min_apsc) / temp_exetime);
    }
    
    for (int i = 0; i < SWARMSIZE; i++) order[i] = i;

    for (int i = 0; i < SWARMSIZE; i++)
    {
        for (int j = i + 1; j < SWARMSIZE; j++)
        {
            if (distance[i] < distance[j])
            {
                float temp_dis = 0;
                int temp_order = 0;

                temp_dis = distance[i];
                distance[i] = distance[j];
                distance[j] = temp_dis;

                temp_order = order[i];
                order[i] = order[j];
                order[j] = temp_order;
            }
        }
    }
}

void ea_crossover_serial(int *rand_cross, int *dp, int num_rand)
{
    for (int i = 0; i < SWARMSIZE; i++)
    {
        for (int j = 0; j < num_rand; j++)
        {
            int temp_change = dp[i * MAXGENE + rand_cross[i * 2 * num_rand +
                j]];
            dp[i * MAXGENE + rand_cross[i * 2 * num_rand + j]] = dp[i * MAXGENE
                + rand_cross[i * 2 * num_rand + j + num_rand]];
            dp[i * MAXGENE + rand_cross[i * 2 * num_rand + j + num_rand]] =
                temp_change;
        }
    }
}

extern "C" void EA_crossover_serial(Swarms* sw_pt, int num_testcase)
{
    //int *d_particle = NULL;
    //int *d_pbest = NULL, *d_speed = NULL;
    //float *d_apsc = NULL, *d_exetime = NULL;
    //size_t ind_size = SWARMSIZE * MAXGENE;

    //CUDA_CALL(hipMalloc((void**)&d_particle, sizeof(int) * ind_size));
    //CUDA_CALL(hipMemset(d_particle, 0, sizeof(int) * ind_size));
    //CUDA_CALL(hipMalloc((void**)&d_pbest, sizeof(int) * ind_size));
    //CUDA_CALL(hipMemset(d_pbest, 0, sizeof(int) * ind_size));
    //CUDA_CALL(hipMalloc((void**)&d_speed, sizeof(int) * ind_size));
    //CUDA_CALL(hipMemset(d_speed, 0, sizeof(int) * ind_size));
    //CUDA_CALL(hipMalloc((void**)&d_apsc, sizeof(float) * SWARMSIZE));
    //CUDA_CALL(hipMemset(d_apsc, 0, sizeof(float) * SWARMSIZE));
    //CUDA_CALL(hipMalloc((void**)&d_exetime, sizeof(float) * SWARMSIZE));
    //CUDA_CALL(hipMemset(d_exetime, 0, sizeof(float) * SWARMSIZE));

    //int *d_gbest = NULL;
    //bool *d_geffic = NULL;
    //size_t g_size = (sw_pt->EAlen) * MAXGENE;
    
    //CUDA_CALL(hipMalloc((void**)&d_gbest, sizeof(int) * g_size));
    //CUDA_CALL(hipMemset(d_gbest, 0, sizeof(int) * g_size));
    //CUDA_CALL(hipMalloc((void**)&d_geffic, sizeof(bool) * sw_pt->EAlen));
    //CUDA_CALL(hipMemset(d_geffic, 0, sizeof(bool) * sw_pt->EAlen));

    //CUDA_CALL(hipMemcpy(d_particle, &(sw_pt->particle[0]), sizeof(int) * ind_size,
    //            hipMemcpyHostToDevice));
    //CUDA_CALL(hipMemcpy(d_pbest, &(sw_pt->pbest[0]), sizeof(int) * ind_size,
    //            hipMemcpyHostToDevice));
    //CUDA_CALL(hipMemcpy(d_speed, &(sw_pt->speed[0]), sizeof(int) * ind_size,
    //            hipMemcpyHostToDevice));
    //CUDA_CALL(hipMemcpy(d_apsc, &(sw_pt->apsc[0]), sizeof(float) * SWARMSIZE,
    //            hipMemcpyHostToDevice));
    //CUDA_CALL(hipMemcpy(d_exetime, &(sw_pt->exetime[0]), sizeof(float) *
    //            SWARMSIZE, hipMemcpyHostToDevice));
    //CUDA_CALL(hipMemcpy(d_gbest, &(sw_pt->EA[0]), sizeof(int) * g_size,
    //            hipMemcpyHostToDevice));
    //CUDA_CALL(hipMemcpy(d_geffic, &(sw_pt->EAefficient[0]), sizeof(bool) *
    //            sw_pt->EAlen, hipMemcpyHostToDevice));

    //int *d_pg = NULL, *d_sp = NULL; //pg: pbest & gbest  sp:speed & pbest
    //int *d_order = NULL;
    //bool *d_mark = NULL;
    //CUDA_CALL(hipMalloc((void**)&d_order, sizeof(int) * SWARMSIZE));
    //CUDA_CALL(hipMemset(d_order, 0,sizeof(int) * SWARMSIZE));
    //CUDA_CALL(hipMalloc((void**)&d_pg, sizeof(int) * ind_size));
    //CUDA_CALL(hipMemset(d_pg, 0, sizeof(int) * ind_size));
    //CUDA_CALL(hipMalloc((void**)&d_sp, sizeof(int) * ind_size));
    //CUDA_CALL(hipMemset(d_sp, 0, sizeof(int) * ind_size));
    //CUDA_CALL(hipMalloc((void**)&d_mark, sizeof(bool) * ind_size));
    //CUDA_CALL(hipMemset(d_mark, 0, sizeof(bool) * ind_size));

    float max_apsc = (float)MIN, min_apsc = (float)MAX;
    float max_exetime = (float)MIN, min_exetime = (float)MAX;
    for (int i = 0; i < SWARMSIZE; i++)
    {
        if (sw_pt->apsc[i] > max_apsc)
        {
            max_apsc = sw_pt->apsc[i];
        }
        if (sw_pt->apsc[i] < min_apsc)
        {
            min_apsc = sw_pt->apsc[i];
        }
        if (sw_pt->exetime[i] > max_exetime)
        {
            max_exetime = sw_pt->exetime[i];
        }
        if (sw_pt->exetime[i] < min_exetime)
        {
            min_exetime = sw_pt->exetime[i];
        }
    }

    //cal_distance_order<<<1, SWARMSIZE>>>(d_order, d_apsc, d_exetime, max_apsc,
    //        min_apsc, max_exetime, min_exetime);
    
    int order[SWARMSIZE];
    for (int i = 0; i < SWARMSIZE; i++) order[i] = 0;

    distance_order_serial(&(order[0]), sw_pt->apsc, sw_pt->exetime, max_apsc,
            min_apsc, max_exetime, min_exetime);

    int ea_count = 0;
    int ea_member[ARCHSIZE];
    for (int i = 0; i < ARCHSIZE; i++) ea_member[i] = 0;

    for (int i = 0; i < sw_pt->EAlen; i++)
    {
        if (sw_pt->EAefficient[i] == 1)
        {
            ea_member[ea_count] = i;
            ea_count += 1;
        }
    }

    int percent = SWARMSIZE / (2 * ea_count);
    int g_order[SWARMSIZE / 2];
    for (int i = 0; i < SWARMSIZE / 2; i++) g_order[i] = 0;
    for (int i = 0; i < ea_count; i++)
    {
        for (int j = 0; j < percent; j++)
        {
            g_order[i * percent + j] = ea_member[i];
        }
    }

    int *dp = NULL;
    dp = (int*)malloc(sizeof(int) * SWARMSIZE * MAXGENE);
    for (int i = 0; i < SWARMSIZE / 2; i++)
    {
        for (int j = 0; j < num_testcase; j++)
        {
            dp[i * MAXGENE + j] = sw_pt->EA[g_order[i] * MAXGENE + j];
        }
    }

    for (int i = SWARMSIZE / 2; i < SWARMSIZE; i++)
    {
        for (int j = 0; j < num_testcase; j++)
        {
            dp[i * MAXGENE + j] = sw_pt->particle[order[i] * MAXGENE + j];
        }
    }

    //int *rand_num = NULL, *d_rand = NULL;
    //rand_num = (int*)malloc(sizeof(int) * SWARMSIZE);
    //CUDA_CALL(hipMalloc((void**)&d_rand, sizeof(int) * SWARMSIZE));
    //CUDA_CALL(hipMemset(d_rand, 0, sizeof(int) * SWARMSIZE));
     
    //srand((unsigned int)time(0));
    //for (int i = 0; i < SWARMSIZE / 2; i++)
    //    rand_num[i] = rand() % sw_pt->eff_num[i];
        //rand_num[i] = rand() % num_testcase;
    //CUDA_CALL(hipMemcpy(d_rand, rand_num, sizeof(int) * SWARMSIZE / 2,
    //            hipMemcpyHostToDevice));
 
    //int blockx = (num_testcase + 512) / 512;
    //dim3 grid(blockx, SWARMSIZE / 2);

    //copy_gbest<<<grid, 512>>>(d_rand, d_gbest, d_pg, d_geffic, num_testcase, 
    //        sw_pt->EAlen);

    //copy_gbest<<<1, SWARMSIZE / 2>>>(d_rand, d_gbest, d_pg, d_geffic, num_testcase, 
    //        sw_pt->EAlen);

    //copy_particle<<<1, SWARMSIZE / 2>>>(d_order, d_particle, d_pg, num_testcase);

    int *rand_cross = NULL; //, *d_rand_cross = NULL;
    int num_rand = (int)((num_testcase * MUTRATE) / 8) * 8;
    if (num_rand > 512) num_rand = 512;

    rand_cross = (int*)malloc(sizeof(int) * num_rand * 2 * SWARMSIZE);
    //CUDA_CALL(hipMalloc((void**)&d_rand_cross, sizeof(int) * num_rand * 2 *
    //            SWARMSIZE));
    
    int *str_mark = (int*)malloc(sizeof(int) * num_testcase);
    //srand((unsigned int)time(NULL)); 
    for (int i = 0; i < SWARMSIZE; i++)
    {
        for (int j = 0; j < num_testcase; j++) str_mark[j] = 0;
        
        srand((unsigned int)time(NULL) * i); 
        for (int j = 0; j < num_rand * 2; j++)
        {
            int temp_rand = rand() % num_testcase;
            if (str_mark[temp_rand] != 0)
            {
                for (int k = 1; k < num_testcase; k += 7)
                {
                    if (temp_rand + k >= num_testcase)
                    {
                        temp_rand = 0;
                    }
                    if (str_mark[temp_rand + k] == 0)
                    {
                        rand_cross[i * num_rand * 2 + j] = temp_rand + k;
                        str_mark[temp_rand + k] += 1;
                        break;
                    }
                    else continue;
                }
            }
            else
            {
                rand_cross[i * num_rand * 2 + j] = temp_rand;
                str_mark[temp_rand] += 1;
            }
        }
    }
    free(str_mark);

    ea_crossover_serial(rand_cross, dp, num_rand);

    for (int i = 0; i < SWARMSIZE; i++)
    {
        for (int j = 0; j < num_testcase; j++)
        {
            sw_pt->particle[i * MAXGENE + j] = dp[i * MAXGENE + j];
        }
    }
    //CUDA_CALL(hipMemcpy(d_rand_cross, rand_cross, sizeof(int) * num_rand * 2 *
    //            SWARMSIZE, hipMemcpyHostToDevice));

    //ea_crossover<<<SWARMSIZE, num_rand, sizeof(int) * num_rand * 2>>>(d_rand_cross, d_pg, num_rand);

    //crossover_g_singlepoint<<<1, SWARMSIZE>>>(d_pg, d_particle, d_gbest, d_geffic, d_mark, d_rand,
    //        num_testcase, sw_pt->EAlen);
    //CUDA_CALL(hipMemcpy(&(sw_pt->particle[0]), d_pg, sizeof(int) * ind_size,
    //            hipMemcpyDeviceToHost));
    /*
    CUDA_CALL(hipMemset(d_mark, 0, sizeof(bool) * ind_size));
    srand((unsigned int)time(0) * 2);
    for (int i = 0; i < SWARMSIZE; i++)
        rand_num[i] = rand() % sw_pt->eff_num[i];
        //rand_num[i] = rand() % num_testcase;
    CUDA_CALL(hipMemcpy(d_rand, rand_num, sizeof(int) * SWARMSIZE,
                hipMemcpyHostToDevice));
    
    crossover_singlepoint<<<1, SWARMSIZE>>>(d_sp, d_speed, d_pg, d_mark, d_rand, num_testcase);
    CUDA_CALL(hipMemcpy(&(sw_pt->speed[0]), d_sp, sizeof(int) * ind_size,
                hipMemcpyDeviceToHost));

    CUDA_CALL(hipMemset(d_mark, 0, sizeof(bool) * ind_size));
    srand((unsigned int)time(0) * 3);
    for (int i = 0; i < SWARMSIZE; i++)
        rand_num[i] = rand() % sw_pt->eff_num[i];
        //rand_num[i] = rand() % num_testcase;
    CUDA_CALL(hipMemcpy(d_rand, rand_num, sizeof(int) * SWARMSIZE,
                hipMemcpyHostToDevice));
    
    crossover_singlepoint<<<1, SWARMSIZE>>>(d_pg, d_particle, d_sp, d_mark, d_rand, num_testcase);
    CUDA_CALL(hipMemcpy(&(sw_pt->particle[0]), d_pg, sizeof(int) * ind_size,
                hipMemcpyDeviceToHost));
    */

    //crossover_g_singlepoint<<<1, SWARMSIZE>>>(d_pg, d_particle, d_gbest, d_geffic, d_mark, d_rand,
    //        num_testcase, sw_pt->EAlen);
    //CUDA_CALL(hipMemcpy(&(sw_pt->particle[0]), d_pg, sizeof(int) * ind_size,
    //            hipMemcpyDeviceToHost));

    //testby-------------
    /*
    int *out = NULL;
    out = (int*)malloc(sizeof(int) * ind_size);
    hipMemcpy(out, d_pg, sizeof(int) * ind_size,hipMemcpyDeviceToHost);
    
    int *check = NULL;
    check = (int*)malloc(sizeof(int) * num_testcase);
    for (int i = 0; i < num_testcase; i++) check[i] = 0;

    for (int i = 0; i < 100; i++)
    {
        for (int j = 0; j < num_testcase; j++) check[j] = 0;

        for (int j = 0; j < num_testcase; j++)
        {
            check[out[j]] += 1;
        }

        for (int j = 0; j < num_testcase; j++)
        {
            if (check[j] != 1)
            {
                cout << "pos: " << i << "  num: " << j << " error: " << check[j] << endl;
            }
        }
    }
    
    cout << endl <<endl;
    cout << "Finish" << endl;
    cout << endl;
    getchar();

    free(out);
    */
    //-------------
    
    //CUDA_CALL(hipFree(d_particle));
    //CUDA_CALL(hipFree(d_pbest));
    //CUDA_CALL(hipFree(d_gbest));
    //CUDA_CALL(hipFree(d_speed));
    //CUDA_CALL(hipFree(d_geffic));
    //CUDA_CALL(hipFree(d_apsc));
    //CUDA_CALL(hipFree(d_exetime));
    //CUDA_CALL(hipFree(d_pg));
    //CUDA_CALL(hipFree(d_sp));
    //CUDA_CALL(hipFree(d_rand));
    //free(rand_num);
    free(rand_cross);
    //CUDA_CALL(hipFree(d_rand_cross));
}
